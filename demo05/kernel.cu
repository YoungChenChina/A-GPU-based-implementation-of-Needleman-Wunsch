#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <cassert>
#include <string>
#include <time.h>

#define GAP -2
#define MATCH 1
#define MISMATCH -1

#define LEFT 'a'
#define TOP 'b'
#define DIA 'c'

#define SEQ_MAX_LEN 480		// SEQ_MAX_LEN >= max length of sequence in data set
#define NUM_BLOCK 16		//block number
#define NUM_STREAM 10		//stream number
#define TASKS 3000000		//number of items from data set to process
#define SKIPS 0				//skip first N items

using namespace std;




__global__
void alignKernel(char *f_seq1, char *f_seq2, int*f_out, char *f_trace, int* f_len_reads, int* f_len_haplotypes, int max_len, int*f_score)
{
	

	char* d_seq1 = f_seq1;
	char* d_seq2 = f_seq2;
	int* e_len_reads = f_len_reads;
	int* e_len_haplotypes = f_len_haplotypes;
	int* d_out = f_out;
	char* trace = f_trace;
	int* d_score = f_score;
	
	//thread indexes
	int thread_id = threadIdx.x + blockIdx.x*blockDim.x;

	//uniformize thread_id to for easier programming
	//each block thread starts from 1
	while (thread_id >= blockDim.x){
		thread_id -= blockDim.x;
		d_seq1 += max_len;
		d_seq2 += max_len;
		e_len_reads += 1;
		e_len_haplotypes += 1;
		d_out += max_len*max_len;
		trace += max_len*max_len;
		d_score += 1;
	}

	//get length of items from reads and haplotypes in specific blocks
	int len_reads = e_len_reads[0];
	int len_haplotypes = e_len_haplotypes[0];
	
	
	int N = 0;
	if (len_reads >len_haplotypes)
		N = len_reads;
	else
		N = len_haplotypes;

	int block_size = N + 1;

	//initialize the first row and the first column of score matrix
	if (thread_id <= N){
		d_out[thread_id] = thread_id * GAP;
		d_out[thread_id *  (len_reads + 1)] = thread_id * GAP;
	}

	//threads synchronization
	__syncthreads();


	int row = 0, col = 0, left = 0, top = 0, dia = 0, max = 0, tra = 0;

	//first half matrix
	for (int i = 1; i <= block_size; i++){

		row = thread_id;
		col = i - thread_id;

		if (thread_id <= i && row > 0 && col > 0 && row <= len_haplotypes && col <= len_reads){

			//get scroes from left grid and top grid
			left = d_out[row*(len_reads + 1) + col - 1] + GAP;
			top = d_out[(row - 1)*(len_reads + 1) + col] + GAP;


			//match score
			if (d_seq2[row - 1] == d_seq1[col - 1]){
				dia = d_out[(row - 1)*(len_reads + 1) + col - 1] + MATCH;
			}

			//mismatch score
			else {				
				dia = d_out[(row - 1)*(len_reads + 1) + col - 1] + MISMATCH;				
			}

			//get final score and trace mark
			if (left > top){
				max = left;
				tra = LEFT;
			}
			else{
				max = top;
				tra = TOP;
			}

			if (dia > max){
				max = dia;
				tra = DIA;
			}
			
			//fill in score matrix and trace matrix
			d_out[row*(len_reads + 1) + col] = max;
			trace[row*(len_reads + 1) + col] = tra;

		}

		__syncthreads();

	}


	//left half matrix ; using the similar process of the first half matrix
	for (int j = 2; j <= block_size; j++){

		row = block_size - 1 - thread_id + j;
		col = thread_id;

		if (thread_id >= j - 1 && thread_id <= block_size && row > 0 && col > 0 && row <= len_haplotypes && col <= len_reads){
			
			left = d_out[row*(len_reads + 1) + col - 1] + GAP;
			top = d_out[(row - 1)*(len_reads + 1) + col] + GAP;
			
			if (d_seq2[row - 1] == d_seq1[col - 1])
				dia = d_out[(row - 1)*(len_reads + 1) + col - 1] + MATCH;
			else dia = d_out[(row - 1)*(len_reads + 1) + col - 1] + MISMATCH;

			if (left > top){
				max = left;
				tra = LEFT;
			}
			else{
				max = top;
				tra = TOP;
			}

			if (dia > max){
				max = dia;
				tra = DIA;
			}

			d_out[row*(len_reads + 1) + col] = max;
			trace[row*(len_reads + 1) + col] = tra;
		}

		__syncthreads();

	}

	//get last score
	if (thread_id == 0){
		d_score[0] = d_out[(len_reads + 1)*(len_haplotypes + 1) - 1];
	}


}


//function to communicate between host and device
void alignArray(char*seq1, char*seq2, int*len_haplotypes, int*len_reads, char*d_seq1, char*d_seq2, int*d_matrix_score, char*d_matrix_trace, int*scores, char*matrix_trace, int*d_len_reads, int*d_len_haplotypes, int max_len, int*d_score){


	//create streams
	hipStream_t stream[NUM_STREAM];
	for (int i = 0; i < NUM_STREAM; i++)
		hipStreamCreate(&stream[i]);

	//run streams
	for (int i = 0; i < NUM_STREAM; i++){

		//copy data from host to device
		hipMemcpyAsync(d_seq1 + i * max_len*NUM_BLOCK, seq1 + i * max_len*NUM_BLOCK, max_len*NUM_BLOCK*sizeof(char), hipMemcpyHostToDevice, stream[i]);
		hipMemcpyAsync(d_seq2 + i * max_len*NUM_BLOCK, seq2 + i * max_len*NUM_BLOCK, max_len*NUM_BLOCK*sizeof(char), hipMemcpyHostToDevice, stream[i]);
		hipMemcpyAsync(d_len_reads + i * NUM_BLOCK, len_reads + i * NUM_BLOCK, NUM_BLOCK*sizeof(int), hipMemcpyHostToDevice, stream[i]);
		hipMemcpyAsync(d_len_haplotypes + i * NUM_BLOCK, len_haplotypes + i * NUM_BLOCK, NUM_BLOCK*sizeof(int), hipMemcpyHostToDevice, stream[i]);
	
		//run kernel
		alignKernel << <NUM_BLOCK, max_len, 0, stream[i] >> >(d_seq1 + i * max_len*NUM_BLOCK, d_seq2 + i * max_len*NUM_BLOCK, d_matrix_score + i * max_len* max_len*NUM_BLOCK, d_matrix_trace + i * max_len* max_len*NUM_BLOCK, d_len_reads + i*NUM_BLOCK, d_len_haplotypes + i*NUM_BLOCK, max_len, d_score + i*NUM_BLOCK);

		//copy back
		hipMemcpyAsync(scores + i *NUM_BLOCK, d_score + i*NUM_BLOCK, NUM_BLOCK*sizeof(int), hipMemcpyDeviceToHost, stream[i]);
		hipMemcpyAsync(matrix_trace + i *max_len* max_len*NUM_BLOCK, d_matrix_trace + i *max_len* max_len*NUM_BLOCK, max_len* max_len*NUM_BLOCK*sizeof(char), hipMemcpyDeviceToHost, stream[i]);
		
	}

	// destroy streams
	for (int i = 0; i < NUM_STREAM; ++i)
		hipStreamDestroy(stream[i]);

}


//big loop to read in data from file
void loop(string *reads, string *haplotypes, int* d_matrix_score, char* d_seq1, char* d_seq2, char* d_matrix_trace, int*d_len_reads, int* d_len_haplotypes, int*d_score, ofstream &outf){
	
	
	int num_data = NUM_BLOCK*NUM_STREAM;		//number of data in one big loop
	int *len_reads = (int*)calloc(num_data, sizeof(int));
	int *len_haplotypes = (int*)calloc(num_data, sizeof(int));

	//get max length of data in one loop for unification malloc
	int max_len = 0;

	for (int i = 0; i < num_data; i++){
		len_reads[i] = reads[i].length();
		len_haplotypes[i] = haplotypes[i].length();
		if (len_reads[i] > max_len)
			max_len = len_reads[i];
		if (len_haplotypes[i] > max_len)
			max_len = len_haplotypes[i];
	}
	max_len += 1;

	//sequences from host to the device
	char*seq1, *seq2;
	hipHostAlloc(&seq1, max_len * num_data* sizeof(char), hipHostMallocDefault);
	hipHostAlloc(&seq2, max_len * num_data* sizeof(char), hipHostMallocDefault);

	//scores and trace matrix 
	int *scores = (int*)calloc(num_data, sizeof(int));
	char *matrix_trace = (char*)calloc(num_data*max_len*max_len, sizeof(char));
	

	//copy items from file to seq1 and seq2
	for (int i = 0; i < num_data; i++){

		strcpy(seq1 + max_len * i, haplotypes[i].c_str());
		strcpy(seq2 + max_len * i, reads[i].c_str());

	}

	//cuda function
	alignArray(seq1, seq2, len_reads, len_haplotypes, d_seq1, d_seq2, d_matrix_score, d_matrix_trace, scores, matrix_trace, d_len_reads, d_len_haplotypes, max_len, d_score);




	//output results

	for (int i = 0; i < num_data; i++){

		//get output sequence
		int row = len_reads[i];
		int col = len_haplotypes[i];
		int t_seq = 0;
		char *trace_seq = (char*)calloc(row + col, sizeof(char));

		//get trace sequence array
		while (row > 0 && col >0){

			trace_seq[t_seq] = matrix_trace[max_len*max_len*i + row*(len_haplotypes[i] + 1) + col];
			if (trace_seq[t_seq] == DIA){
				row--;
				col--;
			}
			else if (trace_seq[t_seq] == LEFT){
				col--;
			}
			else if (trace_seq[t_seq == TOP]){
				row--;
			}
			t_seq++;

		}

		while (row > 0){
			trace_seq[t_seq] = TOP;
			row--;
			t_seq++;
		}
		while (col > 0){
			trace_seq[t_seq] = LEFT;
			col--;
			t_seq++;
		}
		

		int output_len = t_seq;

		char *out_reads = new char[output_len];
		char *out_haplotypes = new char[output_len];


		//output haplotype

		int haplotypes_out = 0;

		for (int t_out = 0; t_out < output_len; t_out++){
			if (trace_seq[output_len - t_out - 1] == DIA){
				
				out_haplotypes[t_out] = seq1[haplotypes_out + i*max_len];
				haplotypes_out++;

			}
			else if (trace_seq[output_len - t_out - 1] == TOP){
				
				out_haplotypes[t_out] = '_';

			}
			else if (trace_seq[output_len - t_out - 1] == LEFT){
				
				out_haplotypes[t_out] = seq1[haplotypes_out + i*max_len];
				haplotypes_out++;

			}
		}

		for (int b = 0; b < output_len; b++){

			outf << out_haplotypes[b];

		}

		outf << endl;

		//output score
		outf << scores[i] << endl;

		//output read
		int read_out = 0;
		for (int t_out = 0; t_out < output_len; t_out++){
			if (trace_seq[output_len - t_out - 1] == DIA){

				out_reads[t_out] = seq2[read_out + i*max_len];
				read_out++;

			}
			else if (trace_seq[output_len - t_out - 1] == TOP){

				out_reads[t_out] = seq2[read_out + i*max_len];
				read_out++;

			}
			else if (trace_seq[output_len - t_out - 1] == LEFT){

				out_reads[t_out] = '_';

			}
		}

		for (int b = 0; b < output_len; b++){

			outf << out_reads[b];

		}

		outf << endl;
		outf << endl;


		delete out_reads;
		delete out_haplotypes;
		free(trace_seq);


		//display results in screen
		//cout << endl;
		//cout << "****************************************************************************************************" << endl;

		//for (int x = 0; x < len_reads[i] + 1; x++){
		//	for (int y = 0; y < len_haplotypes[i] + 1; y++){
		//		cout << matrix_trace[i*max_len*max_len + x*(len_haplotypes[i] + 1) + y];
		//		cout << "\t";
		//	}
		//	cout << endl;
		//}

		//cout << "x=" << i << endl;	//number of rows in files

		//cout << "     reads:" << reads[i] << endl;
		//cout << "haplotypes:" << haplotypes[i] << endl;
		//cout << "length of reads:" << len_reads[i] << endl;
		//cout << "length of haplotypes:" << len_haplotypes[i] << endl;

		//cout << "score" << i << ":" << scores[i] << endl;

		//cout << "trace_seq:";
		//for (int x = 0; x < t_seq; x++){
		//	cout << trace_seq[x];
		//}
		//cout << endl;

		//cout << "row:" << row << endl;
		//cout << "col:" << col << endl;

		//cout << "len_output_seq:" << t_seq<<endl;

		//cout << "haplotypes_out:";
		//for (int b = 0; b < output_len; b++){
		//	cout << out_haplotypes[b];
		//}
		//cout<<endl;

		//cout << "      read_out:";
		//for (int b = 0; b < output_len; b++){
		//	cout << out_reads[b];
		//}
		//cout<<endl;

		//cout << "****************************************************************************************************" << endl;

		//cout << endl;


	}

	
	//free memory

	free(len_reads);
	free(len_haplotypes);
	free(scores);
	free(matrix_trace);
	
	hipHostFree(seq1);
	hipHostFree(seq2);


}





int main()
{
	//open files
	ifstream readsfile("H:/Hust/������/��׼ҽ����Ϣ����/project/GPU/����/reads.txt");						//reads file
	ifstream haplotypesfile("H:/Hust/������/��׼ҽ����Ϣ����/project/GPU/����/haplotypes.txt");			//haplotypes file
	ofstream outf("H:/Hust/������/��׼ҽ����Ϣ����/project/GPU/����/gpu_output/AlignResults.txt");			//output file
	ofstream timef("H:/Hust/������/��׼ҽ����Ϣ����/project/GPU/����/gpu_output/demo05/time.txt", ios::app);	//time record file


	//ifstream readsfile("H:/Hust/������/��׼ҽ����Ϣ����/project/GPU/��������/reads.txt");
	//ifstream haplotypesfile("H:/Hust/������/��׼ҽ����Ϣ����/project/GPU/��������/haplotypes.txt");
	//ofstream outf("H:/Hust/������/��׼ҽ����Ϣ����/project/GPU/��������/AlignResults.txt");
	//ofstream timef("H:/Hust/������/��׼ҽ����Ϣ����/project/GPU/��������/gpu_output/time.txt", ios::app);


	//if fail to open files, break
	assert(readsfile.is_open());
	assert(haplotypesfile.is_open());
	assert(outf.is_open);
	assert(timef.is_open);


	string skip_reads, skip_hap;
	for (int c = 0; c < SKIPS; c++){
		getline(readsfile, skip_reads);
		getline(haplotypesfile, skip_hap);
	}

	//program running time
	double start, stop, durationTime;
	start = clock();
	
	
	int loop_main = int(TASKS / (NUM_STREAM*NUM_BLOCK));



	//malloc GPU

	int *d_matrix_score = 0;
	char *d_seq1 = 0;
	char *d_seq2 = 0;
	char *d_matrix_trace = 0;
	int *d_len_reads = 0;
	int *d_len_haplotypes = 0;
	int *d_score = 0;

	hipMalloc(&d_matrix_score, (SEQ_MAX_LEN + 1) * (SEQ_MAX_LEN + 1) * NUM_BLOCK* NUM_STREAM * sizeof(int));
	hipMalloc(&d_seq1, (SEQ_MAX_LEN + 1) * NUM_BLOCK * NUM_STREAM * sizeof(char));
	hipMalloc(&d_seq2, (SEQ_MAX_LEN + 1) * NUM_BLOCK * NUM_STREAM * sizeof(char));
	hipMalloc(&d_matrix_trace, (SEQ_MAX_LEN + 1) * (SEQ_MAX_LEN + 1) * NUM_BLOCK * NUM_STREAM * sizeof(char));
	hipMalloc(&d_len_reads,  NUM_BLOCK * NUM_STREAM * sizeof(int));
	hipMalloc(&d_len_haplotypes, NUM_BLOCK * NUM_STREAM * sizeof(int));
	hipMalloc(&d_score, NUM_BLOCK * NUM_STREAM * sizeof(int));



	for (int x = 0; x < loop_main; x++){

		string reads[NUM_STREAM*NUM_BLOCK];
		string haplotypes[NUM_STREAM*NUM_BLOCK];
		for (int a = 0; a < NUM_STREAM*NUM_BLOCK; a++){
			getline(readsfile, reads[a]);
			getline(haplotypesfile, haplotypes[a]);
		}


		//show rate of progress
		cout << "loop:" << x+1 << endl;
		system("cls");
		
		//strat one loop to process NUM_STREAM*NUM_BLOCK items at the same time
		loop(reads, haplotypes, d_matrix_score, d_seq1, d_seq2, d_matrix_trace, d_len_reads, d_len_haplotypes, d_score, outf);


	}

	stop = clock();

	durationTime = ((double)(stop - start)) / CLK_TCK;
	cout << "time:" << durationTime << endl;


	//free memory
	hipFree(d_matrix_score);
	hipFree(d_seq1);
	hipFree(d_seq2);
	hipFree(d_matrix_trace);
	hipFree(d_len_reads);
	hipFree(d_len_haplotypes);
	hipFree(d_score);

	//output time to time record file
	timef << endl <<TASKS << ":" << durationTime;

	//close files
	readsfile.close();
	haplotypesfile.close();
	outf.close();
	timef.close();
	return 0;


}






